#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>

#include "utility.cuh"

// m=n=k=4096
// kernel cost: 56.7ms
// GFLOPs: 2416.2
__global__ void sgemm_naive_intuitive(int m, int n, int k, float alpha,
                                      const float *A, const float *B,
                                      float beta, float *C) {
  const uint idx =
      blockIdx.z *
          (gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x) +
      blockIdx.y * (gridDim.x * blockDim.z * blockDim.y * blockDim.x) +
      blockIdx.x * (blockDim.z * blockDim.y * blockDim.x) +
      threadIdx.z * (blockDim.y * blockDim.x) + threadIdx.y * blockDim.x +
      threadIdx.x;

  if (idx < m * n) {
    const uint x = idx / n;
    const uint y = idx % n;

    float tmp = 0.0;

    for (int i = 0; i < k; ++i) {
      tmp += A[x * k + i] * B[i * n + y];
    }

    C[x * n + y] = alpha * tmp + beta * C[x * n + y];
  }
}

// kernel cost: 32.7ms
__global__ void sgemm_naive_transpose(int m, int n, int k, float alpha,
                                      const float *A, const float *B,
                                      float beta, float *C) {
  const uint x = blockIdx.y * blockDim.y + threadIdx.y;
  const uint y = blockIdx.x * blockDim.x + threadIdx.x;

  if (x < m && y < n) {
    float tmp = 0.0;

    for (int i = 0; i < k; ++i) {
      tmp += A[x * k + i] * B[i * n + y];
    }

    C[x * n + y] = alpha * tmp + beta * C[x * n + y];
  }
}

// kernel cost: 38.7ms
template <const uint BLOCKSIZE>
__global__ void sgemm_global_mem_coalesce(int m, int n, int k, float alpha,
                                          const float *A, const float *B,
                                          float beta, float *C) {
  const uint x = blockIdx.x * BLOCKSIZE + threadIdx.x / BLOCKSIZE;
  const uint y = blockIdx.y * BLOCKSIZE + threadIdx.x % BLOCKSIZE;

  if (x < m && y < n) {
    float tmp = 0.0;

    for (int i = 0; i < k; ++i) {
      tmp += A[x * k + i] * B[i * n + y];
    }

    C[x * n + y] = alpha * tmp + beta * C[x * n + y];
  }
}

// m=n=k=4096
// kernel cost: 249.7ms
// GFLOPs: 548.8
__global__ void sgemm_naive(int m, int n, int k, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < m && y < n) {
    float tmp = 0.0;

    for (int i = 0; i < k; ++i) {
      tmp += A[x * k + i] * B[i * n + y];
    }

    C[x * n + y] = alpha * tmp + beta * C[x * n + y];
  }
}

int get_array_diff(std::vector<float> &v_a, std::vector<float> &v_b) {
  int diff = 0;
  for (int i = 0; i < v_a.size(); ++i) {
    diff += abs(v_a[i] - v_b[i]);
  }
  return diff;
}

int main() {
  constexpr int m = 1024 * 4;
  constexpr int n = 1024 * 4;
  constexpr int k = 1024 * 4;
  std::vector<float> v_a(m * k, 1.0);
  std::vector<float> v_b(k * n, 1.0);
  std::vector<float> v_c1(m * n, 1.0);
  std::vector<float> v_c2(m * n, 1.0);

  dim3 gridDim((m + 32 - 1) / 32, (n + 32 - 1) / 32, 1);
  dim3 blockDim(32, 32, 1);

  float *a;
  float *b;
  float *c1;
  float *c2;

  util::prepareRandomNumbersCpuGpu(m * k * sizeof(float), v_a, &a);
  util::prepareRandomNumbersCpuGpu(k * n * sizeof(float), v_b, &b);
  util::prepareRandomNumbersCpuGpu(m * n * sizeof(float), v_c1, &c1);
  util::prepareRandomNumbersCpuGpu(m * n * sizeof(float), v_c2, &c2);

  hipEvent_t start, compute_end, copy_end;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&compute_end));
  CHECK_CUDA(hipEventCreate(&copy_end));

  hipEventRecord(start);
  sgemm_naive<<<gridDim, blockDim>>>(m, n, k, 2.0, a, b, 0.0, c1);
  sgemm_global_mem_coalesce<32>
      <<<gridDim, 32 * 32>>>(m, n, k, 2.0, a, b, 0.0, c2);

  // sgemm_naive_transpose<<<gridDim, blockDim>>>(m, n, k, 2.0, a, b, 0.0, c2);
  // sgemm_naive_intuitive<<<gridDim, blockDim>>>(m, n, k, 2.0, a, b, 0.0, c2);
  hipEventRecord(compute_end);

  hipMemcpy(v_c1.data(), c1, m * n * sizeof(float), hipMemcpyDeviceToHost);

  hipMemcpy(v_c2.data(), c2, m * n * sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(copy_end);

  auto err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  float time_compute, time_copy;
  hipEventElapsedTime(&time_compute, start, compute_end);
  hipEventElapsedTime(&time_copy, compute_end, copy_end);

  std::cout << "Time for computation: " << time_compute << " ms" << std::endl;
  std::cout << "Time for copy: " << time_copy << " ms" << std::endl;

  auto diff = get_array_diff(v_c1, v_c2);
  std::cout << "Difference between results: " << diff << std::endl;

  return 0;
}
